#include "hip/hip_runtime.h"
/*-----------------------------------------------------------------------------
   Name: histogramTRISH.cu
   Desc: Implements 256-way binning histogram algorithm on GPU
   
   Disclaimer:
      This software is distributed in the hope that it will be useful,
      but WITHOUT ANY WARRANTY; without even the implied warranty of
      MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
-----------------------------------------------------------------------------*/

/*-----------------------------------------------------------------------------
  Includes
-----------------------------------------------------------------------------*/

// System Includes
#include <assert.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// CUDA Includes
#include <cutil_inline.h>

// Local Includes
#include "histogram_common.h"


/*-----------------------------------------------------------------------------
  Helper Templates
-----------------------------------------------------------------------------*/

/*---------------------------------------------------------
  Name:   SetArray_BlockSeq
  Desc:   Sets elements in array to specified value
  Note:   Uses "Block Sequential" access pattern
 ---------------------------------------------------------*/

template <  typename valT,		// Underlying value type
            uint BlockSize,    // ThreadPerBlock
            uint nSafePasses,  // Number of safe passes
            uint nLeftOver,    // Number of left over elements
            uint maxSize >     // Max Size of array
__device__ __forceinline__
void SetArray_BlockSeq
( 
   valT * basePtr,      // IN/OUT - array to set to 'set' value
   valT   toSet         // IN - value to set array elements 'to'
) 
{
   // Get 'per thread' pointer
   valT * setPtr = basePtr + threadIdx.x;

		// Initialize as many elements as we
		// safely can with no range checking
	if (nSafePasses >=  1u) { setPtr[( 0u * BlockSize)] = toSet; }
	if (nSafePasses >=  2u) { setPtr[( 1u * BlockSize)] = toSet; }
	if (nSafePasses >=  3u) { setPtr[( 2u * BlockSize)] = toSet; }
	if (nSafePasses >=  4u) { setPtr[( 3u * BlockSize)] = toSet; }
	if (nSafePasses >=  5u) { setPtr[( 4u * BlockSize)] = toSet; }
	if (nSafePasses >=  6u) { setPtr[( 5u * BlockSize)] = toSet; }
	if (nSafePasses >=  7u) { setPtr[( 6u * BlockSize)] = toSet; }
	if (nSafePasses >=  8u) { setPtr[( 7u * BlockSize)] = toSet; }
	if (nSafePasses >=  9u) { setPtr[( 8u * BlockSize)] = toSet; }
	if (nSafePasses >= 10u) { setPtr[( 9u * BlockSize)] = toSet; }
	if (nSafePasses >= 11u) { setPtr[(10u * BlockSize)] = toSet; }
	if (nSafePasses >= 12u) { setPtr[(11u * BlockSize)] = toSet; }
	if (nSafePasses >= 13u) { setPtr[(12u * BlockSize)] = toSet; }
	if (nSafePasses >= 14u) { setPtr[(13u * BlockSize)] = toSet; }
	if (nSafePasses >= 15u) { setPtr[(14u * BlockSize)] = toSet; }
	if (nSafePasses >= 16u) { setPtr[(15u * BlockSize)] = toSet; }
	if (nSafePasses >= 17u) { setPtr[(16u * BlockSize)] = toSet; }
	if (nSafePasses >= 18u) { setPtr[(17u * BlockSize)] = toSet; }
	if (nSafePasses >= 19u) { setPtr[(18u * BlockSize)] = toSet; }
	if (nSafePasses >= 20u) { setPtr[(19u * BlockSize)] = toSet; }
	if (nSafePasses >= 21u) { setPtr[(20u * BlockSize)] = toSet; }
	if (nSafePasses >= 22u) { setPtr[(21u * BlockSize)] = toSet; }
	if (nSafePasses >= 23u) { setPtr[(22u * BlockSize)] = toSet; }
	if (nSafePasses >= 24u) { setPtr[(23u * BlockSize)] = toSet; }
	if (nSafePasses >= 25u) { setPtr[(24u * BlockSize)] = toSet; }
	if (nSafePasses >= 26u) { setPtr[(25u * BlockSize)] = toSet; }
	if (nSafePasses >= 27u) { setPtr[(26u * BlockSize)] = toSet; }
	if (nSafePasses >= 28u) { setPtr[(27u * BlockSize)] = toSet; }
	if (nSafePasses >= 29u) { setPtr[(28u * BlockSize)] = toSet; }
	if (nSafePasses >= 30u) { setPtr[(29u * BlockSize)] = toSet; }
	if (nSafePasses >= 31u) { setPtr[(30u * BlockSize)] = toSet; }
	if (nSafePasses >= 32u) { setPtr[(31u * BlockSize)] = toSet; }
	if (nSafePasses >= 33u) { setPtr[(32u * BlockSize)] = toSet; }
	if (nSafePasses >= 34u) { setPtr[(33u * BlockSize)] = toSet; }
	if (nSafePasses >= 35u) { setPtr[(34u * BlockSize)] = toSet; }
	if (nSafePasses >= 36u) { setPtr[(35u * BlockSize)] = toSet; }
	if (nSafePasses >= 37u) { setPtr[(36u * BlockSize)] = toSet; }
	if (nSafePasses >= 38u) { setPtr[(37u * BlockSize)] = toSet; }
	if (nSafePasses >= 39u) { setPtr[(38u * BlockSize)] = toSet; }
	if (nSafePasses >= 40u) { setPtr[(39u * BlockSize)] = toSet; }
	if (nSafePasses >= 41u) { setPtr[(40u * BlockSize)] = toSet; }
	if (nSafePasses >= 42u) { setPtr[(41u * BlockSize)] = toSet; }
	if (nSafePasses >= 43u) { setPtr[(42u * BlockSize)] = toSet; }
	if (nSafePasses >= 44u) { setPtr[(43u * BlockSize)] = toSet; }
	if (nSafePasses >= 45u) { setPtr[(44u * BlockSize)] = toSet; }
	if (nSafePasses >= 46u) { setPtr[(45u * BlockSize)] = toSet; }
	if (nSafePasses >= 47u) { setPtr[(46u * BlockSize)] = toSet; }
	if (nSafePasses >= 48u) { setPtr[(47u * BlockSize)] = toSet; }
	if (nSafePasses >= 49u) { setPtr[(48u * BlockSize)] = toSet; }
	if (nSafePasses >= 50u) { setPtr[(49u * BlockSize)] = toSet; }
	if (nSafePasses >= 51u) { setPtr[(50u * BlockSize)] = toSet; }
	if (nSafePasses >= 52u) { setPtr[(51u * BlockSize)] = toSet; }
	if (nSafePasses >= 53u) { setPtr[(52u * BlockSize)] = toSet; }
	if (nSafePasses >= 54u) { setPtr[(53u * BlockSize)] = toSet; }
	if (nSafePasses >= 55u) { setPtr[(54u * BlockSize)] = toSet; }
	if (nSafePasses >= 56u) { setPtr[(55u * BlockSize)] = toSet; }
	if (nSafePasses >= 57u) { setPtr[(56u * BlockSize)] = toSet; }
	if (nSafePasses >= 58u) { setPtr[(57u * BlockSize)] = toSet; }
	if (nSafePasses >= 59u) { setPtr[(58u * BlockSize)] = toSet; }
	if (nSafePasses >= 60u) { setPtr[(59u * BlockSize)] = toSet; }
	if (nSafePasses >= 61u) { setPtr[(60u * BlockSize)] = toSet; }
	if (nSafePasses >= 62u) { setPtr[(61u * BlockSize)] = toSet; }
	if (nSafePasses >= 63u) { setPtr[(62u * BlockSize)] = toSet; }
	if (nSafePasses >= 64u) { setPtr[(63u * BlockSize)] = toSet; }
	if (nSafePasses >= 65u) { setPtr[(64u * BlockSize)] = toSet; }
	if (nSafePasses >= 66u) { setPtr[(65u * BlockSize)] = toSet; }

	// Set any 'left over' values with range checking
	if (nLeftOver > 0u)
	{ 
		uint idx = (nSafePasses * BlockSize) + threadIdx.x;
		if (idx < maxSize)
		{
			basePtr[idx] = toSet;
		}
	}
}


/*---------------------------------------------------------
  Name:   SetArray_WarpSeq
  Desc:   Sets elements in array to specified value
  Note:   Uses "Warp Sequential" access pattern
 ---------------------------------------------------------*/

template <  typename valT,		// Underlying value type
            uint WarpSize,     // Threads per Warp
            uint nSafePasses,  // Number of safe passes (warps per subsection)
            uint nLeftOver,    // Number of left over elements
            uint maxSize >     // Max Size of array
__device__ __forceinline__
void SetArray_WarpSeq
( 
   valT * basePtr,      // IN/OUT - array to set to 'set' value
   valT   toSet,        // IN - value to set array elements 'to'
   uint    startIdx      // starting index for this thread
) 
{
   // Get 'per thread' pointer
   valT * setPtr  = &basePtr[startIdx];

		// Initialize as many elements as we
		// safely can with no range checking
	if (nSafePasses >=  1u) { setPtr[( 0u * WarpSize)] = toSet; }
	if (nSafePasses >=  2u) { setPtr[( 1u * WarpSize)] = toSet; }
	if (nSafePasses >=  3u) { setPtr[( 2u * WarpSize)] = toSet; }
	if (nSafePasses >=  4u) { setPtr[( 3u * WarpSize)] = toSet; }
	if (nSafePasses >=  5u) { setPtr[( 4u * WarpSize)] = toSet; }
	if (nSafePasses >=  6u) { setPtr[( 5u * WarpSize)] = toSet; }
	if (nSafePasses >=  7u) { setPtr[( 6u * WarpSize)] = toSet; }
	if (nSafePasses >=  8u) { setPtr[( 7u * WarpSize)] = toSet; }
	if (nSafePasses >=  9u) { setPtr[( 8u * WarpSize)] = toSet; }
	if (nSafePasses >= 10u) { setPtr[( 9u * WarpSize)] = toSet; }
	if (nSafePasses >= 11u) { setPtr[(10u * WarpSize)] = toSet; }
	if (nSafePasses >= 12u) { setPtr[(11u * WarpSize)] = toSet; }
	if (nSafePasses >= 13u) { setPtr[(12u * WarpSize)] = toSet; }
	if (nSafePasses >= 14u) { setPtr[(13u * WarpSize)] = toSet; }
	if (nSafePasses >= 15u) { setPtr[(14u * WarpSize)] = toSet; }
	if (nSafePasses >= 16u) { setPtr[(15u * WarpSize)] = toSet; }
	if (nSafePasses >= 17u) { setPtr[(16u * WarpSize)] = toSet; }
	if (nSafePasses >= 18u) { setPtr[(17u * WarpSize)] = toSet; }
	if (nSafePasses >= 19u) { setPtr[(18u * WarpSize)] = toSet; }
	if (nSafePasses >= 20u) { setPtr[(19u * WarpSize)] = toSet; }
	if (nSafePasses >= 21u) { setPtr[(20u * WarpSize)] = toSet; }
	if (nSafePasses >= 22u) { setPtr[(21u * WarpSize)] = toSet; }
	if (nSafePasses >= 23u) { setPtr[(22u * WarpSize)] = toSet; }
	if (nSafePasses >= 24u) { setPtr[(23u * WarpSize)] = toSet; }
	if (nSafePasses >= 25u) { setPtr[(24u * WarpSize)] = toSet; }
	if (nSafePasses >= 26u) { setPtr[(25u * WarpSize)] = toSet; }
	if (nSafePasses >= 27u) { setPtr[(26u * WarpSize)] = toSet; }
	if (nSafePasses >= 28u) { setPtr[(27u * WarpSize)] = toSet; }
	if (nSafePasses >= 29u) { setPtr[(28u * WarpSize)] = toSet; }
	if (nSafePasses >= 30u) { setPtr[(29u * WarpSize)] = toSet; }
	if (nSafePasses >= 31u) { setPtr[(30u * WarpSize)] = toSet; }
	if (nSafePasses >= 32u) { setPtr[(31u * WarpSize)] = toSet; }
	if (nSafePasses >= 33u) { setPtr[(32u * WarpSize)] = toSet; }
	if (nSafePasses >= 34u) { setPtr[(33u * WarpSize)] = toSet; }
	if (nSafePasses >= 35u) { setPtr[(34u * WarpSize)] = toSet; }
	if (nSafePasses >= 36u) { setPtr[(35u * WarpSize)] = toSet; }
	if (nSafePasses >= 37u) { setPtr[(36u * WarpSize)] = toSet; }
	if (nSafePasses >= 38u) { setPtr[(37u * WarpSize)] = toSet; }
	if (nSafePasses >= 39u) { setPtr[(38u * WarpSize)] = toSet; }
	if (nSafePasses >= 40u) { setPtr[(39u * WarpSize)] = toSet; }
	if (nSafePasses >= 41u) { setPtr[(40u * WarpSize)] = toSet; }
	if (nSafePasses >= 42u) { setPtr[(41u * WarpSize)] = toSet; }
	if (nSafePasses >= 43u) { setPtr[(42u * WarpSize)] = toSet; }
	if (nSafePasses >= 44u) { setPtr[(43u * WarpSize)] = toSet; }
	if (nSafePasses >= 45u) { setPtr[(44u * WarpSize)] = toSet; }
	if (nSafePasses >= 46u) { setPtr[(45u * WarpSize)] = toSet; }
	if (nSafePasses >= 47u) { setPtr[(46u * WarpSize)] = toSet; }
	if (nSafePasses >= 48u) { setPtr[(47u * WarpSize)] = toSet; }
	if (nSafePasses >= 49u) { setPtr[(48u * WarpSize)] = toSet; }
	if (nSafePasses >= 50u) { setPtr[(49u * WarpSize)] = toSet; }
	if (nSafePasses >= 51u) { setPtr[(50u * WarpSize)] = toSet; }
	if (nSafePasses >= 52u) { setPtr[(51u * WarpSize)] = toSet; }
	if (nSafePasses >= 53u) { setPtr[(52u * WarpSize)] = toSet; }
	if (nSafePasses >= 54u) { setPtr[(53u * WarpSize)] = toSet; }
	if (nSafePasses >= 55u) { setPtr[(54u * WarpSize)] = toSet; }
	if (nSafePasses >= 56u) { setPtr[(55u * WarpSize)] = toSet; }
	if (nSafePasses >= 57u) { setPtr[(56u * WarpSize)] = toSet; }
	if (nSafePasses >= 58u) { setPtr[(57u * WarpSize)] = toSet; }
	if (nSafePasses >= 59u) { setPtr[(58u * WarpSize)] = toSet; }
	if (nSafePasses >= 60u) { setPtr[(59u * WarpSize)] = toSet; }
	if (nSafePasses >= 61u) { setPtr[(60u * WarpSize)] = toSet; }
	if (nSafePasses >= 62u) { setPtr[(61u * WarpSize)] = toSet; }
	if (nSafePasses >= 63u) { setPtr[(62u * WarpSize)] = toSet; }
	if (nSafePasses >= 64u) { setPtr[(63u * WarpSize)] = toSet; }
	if (nSafePasses >= 65u) { setPtr[(64u * WarpSize)] = toSet; }
	if (nSafePasses >= 66u) { setPtr[(65u * WarpSize)] = toSet; }

	// Set any 'left over' values with range checking
	if (nLeftOver > 0u)
	{ 
		uint idx = startIdx + (nSafePasses * WarpSize);
		if (idx < maxSize)
		{
			basePtr[idx] = toSet;
		}
	}
}


/*-------------------------------------------------------------------
  Name:   Bin4_None
  Desc:   *NO* Range check on binning
         Bins 1st & 3rd bytes in value
         Bins 2nd & 4th bytes in value
 ------------------------------------------------------------------*/

template <uint BlockSize>
__device__ __forceinline__
void Bin4_None
( 
   uint * cntPtr, // OUT - count array (to store bin results in)
   uint   val32   // IN  - input 'value' to count
) 
{
   //const uint maskRow13 = 0x003F003Fu;  // Mask for 1st and 3rd lanes
   const uint maskRow13 = 0x0FC00FC0u;
   const uint maskCol   = 0x03030303u;  // Mask for columns

   // Get 'LaneRows' from bins
      // [0..63] = [0..255]/4
   //uint laneRow13 = val32 >> 2u;     // Divide by 4
   //uint laneRow24 = val32 >> 10u;    // Shift by 8, divide by 4

   uint laneRow13 = val32 << 4u;
   uint laneRow24 = val32 >> 4u;

   // Get Lane Column from bins
      // [0..3] = bin [0..255] % 4
   uint laneCol = val32 & maskCol;

      // Mask off 'laneRows' to avoid extra info
   uint LI_13 = laneRow13 & maskRow13;  // get lanes for 1 & 3 bins
   uint LI_24 = laneRow24 & maskRow13;  // get lanes for 2 & 4 bins

   // Get local indices
   //uint LI_13 = laneRow13 * BlockSize;
   //uint LI_24 = laneRow24 * BlockSize;

   // Get Shifts [0,8,16,24] = [0,1,2,3]*8
   uint shift = laneCol << 3u;

   // Get local indices
   uint LI_4 = LI_24 >> 16u;
   uint LI_3 = LI_13 >> 16u;
   uint LI_2 = LI_24 & 0xFFFFu;
   uint LI_1 = LI_13 & 0xFFFFu;

   uint s4 = (shift >> 24u);
   uint s3 = (shift >> 16u);
   uint s2 = (shift >>  8u);
   uint s1 = shift & 0xFFu;

   s3 = s3 & 0xFFu;
   s2 = s2 & 0xFFu;

   uint inc4 = 1u << s4;
   uint inc3 = 1u << s3;
   uint inc2 = 1u << s2;
   uint inc1 = 1u << s1;
   
   uint oldCnt, newCnt;

   // Increment 4th bin
   oldCnt = cntPtr[LI_4];
   newCnt = oldCnt + inc4;
   cntPtr[LI_4] = newCnt;

   // Increment 3rd bin
   oldCnt = cntPtr[LI_3];
   newCnt = oldCnt + inc3;
   cntPtr[LI_3] = newCnt;

   // Increment 2nd bin
   oldCnt = cntPtr[LI_2];
   newCnt = oldCnt + inc2;
   cntPtr[LI_2] = newCnt;

   // Increment 1st bin
   oldCnt = cntPtr[LI_1];
   newCnt = oldCnt + inc1;
   cntPtr[LI_1] = newCnt;
}


/*-------------------------------------------------------------------
  Name:   SS_Sums_4_Next_V1
  Desc:   Serial scan on next 4 elements in seq [0..3]
 ------------------------------------------------------------------*/

template < uint BlockSize,     // Threads per block
           uint BlockMask >    // Block Mask
__device__ __forceinline__
void SS_Sums_4_Next_V1
( 
   uint & sum1,     // OUT - sum1 .. sum4 (as singletons)
   uint & sum2,
   uint & sum3,
   uint & sum4,
   uint * cntPtr,   // IN  - 'per thread' counts <horizontal row> to sum up
   uint   baseIdx
) 
{
   // wrap = (idx + [0..3]) % BlockSize
   uint idx1, idx2, idx3, idx4;
   idx1 = baseIdx + 0u;
   idx2 = baseIdx + 1u;
   idx3 = baseIdx + 2u;
   idx4 = baseIdx + 3u;

   uint wrap1, wrap2, wrap3, wrap4;
   wrap1 = idx1 & BlockMask;
   wrap2 = idx2 & BlockMask;
   wrap3 = idx3 & BlockMask;
   wrap4 = idx4 & BlockMask;

   //-
   // Grab 4 elements in seq [0..3]
   //-

   uint lane1, lane2, lane3, lane4;
   lane1 = cntPtr[wrap1];
   lane2 = cntPtr[wrap2];
   lane3 = cntPtr[wrap3];
   lane4 = cntPtr[wrap4];


   //-
   // Zero out sequence [0..3]
   //-

   cntPtr[wrap1] = 0u;
   cntPtr[wrap2] = 0u;
   cntPtr[wrap3] = 0u;
   cntPtr[wrap4] = 0u;


   //-
   // Accumulate all 4 groups in each lane
   //-

   //-
   // Initialize sums from 1st lane (of 4 groups)
   //-
   uint s3 = lane1 >> 16u;     // 3rd bin (of 4) in lane
   uint s2 = lane1 >>  8u;     // 2nd bin (of 4) in lane

   uint cnt4 = lane1 >> 24u;
   uint cnt3 = s3 & 0xFFu;
   uint cnt2 = s2 & 0xFFu;
   uint cnt1 = lane1 & 0xFFu;

   sum4 = sum4 + cnt4;
   sum3 = sum3 + cnt3;
   sum2 = sum2 + cnt2;
   sum1 = sum1 + cnt1;


   //-
   // Accumulate sums from 2nd lane (of 4 groups)
   //-

   s3 = lane2 >> 16u;     // 3rd bin (of 4) in lane
   s2 = lane2 >>  8u;     // 2nd bin (of 4) in lane

   cnt4 = lane2 >> 24u;
   cnt3 = s3 & 0xFFu;
   cnt2 = s2 & 0xFFu;
   cnt1 = lane2 & 0xFFu;

   sum4 = sum4 + cnt4;
   sum3 = sum3 + cnt3;
   sum2 = sum2 + cnt2;
   sum1 = sum1 + cnt1;


   //-
   // Accumulate sums from 3rd lane (of 4 groups)
   //-

   s3 = lane3 >> 16u;     // 3rd bin (of 4) in lane
   s2 = lane3 >>  8u;     // 2nd bin (of 4) in lane

   cnt4 = lane3 >> 24u;
   cnt3 = s3 & 0xFFu;
   cnt2 = s2 & 0xFFu;
   cnt1 = lane3 & 0xFFu;

   sum4 = sum4 + cnt4;
   sum3 = sum3 + cnt3;
   sum2 = sum2 + cnt2;
   sum1 = sum1 + cnt1;


   //-
   // Accumulate sums from 4th lane (of 4 groups)
   //-

   s3 = lane4 >> 16u;     // 3rd bin (of 4) in lane
   s2 = lane4 >>  8u;     // 2nd bin (of 4) in lane

   cnt4 = lane4 >> 24u;
   cnt3 = s3 & 0xFFu;
   cnt2 = s2 & 0xFFu;
   cnt1 = lane4 & 0xFFu;

   sum4 = sum4 + cnt4;
   sum3 = sum3 + cnt3;
   sum2 = sum2 + cnt2;
   sum1 = sum1 + cnt1;
}


/*-------------------------------------------------------------------
  Name:   SS_Sums_4_Next_V2
  Desc:   Serial scan on next 4 elements in seq [0..3]
 ------------------------------------------------------------------*/

template < uint BlockSize,     // Threads Per Block
           uint BlockMask >    // Block Mask
__device__ __forceinline__
void SS_Sums_4_Next_V2
( 
   uint & sum13,    // OUT - sum1 .. sum4 (as pairs)
   uint & sum24,
   uint * cntPtr,   // IN  - 'per thread' counts <horizontal row> to sum up
   uint   baseIdx
) 
{
   // wrap = (idx + [0..3]) % BlockSize
   uint idx1, idx2, idx3, idx4;
   idx1 = baseIdx + 0u;
   idx2 = baseIdx + 1u;
   idx3 = baseIdx + 2u;
   idx4 = baseIdx + 3u;

   uint wrap1, wrap2, wrap3, wrap4;
   wrap1 = idx1 & BlockMask;
   wrap2 = idx2 & BlockMask;
   wrap3 = idx3 & BlockMask;
   wrap4 = idx4 & BlockMask;

   //-
   // Grab 4 elements in seq [0..3]
   //-

   uint lane1, lane2, lane3, lane4;
   lane1 = cntPtr[wrap1];
   lane2 = cntPtr[wrap2];
   lane3 = cntPtr[wrap3];
   lane4 = cntPtr[wrap4];


   //-
   // Zero out sequence [0..3]
   //-

   cntPtr[wrap1] = 0u;
   cntPtr[wrap2] = 0u;
   cntPtr[wrap3] = 0u;
   cntPtr[wrap4] = 0u;


   //-
   // Accumulate all 4 groups in each lane
   //-

   //-
   // Initialize sums from 1st lane (of 4 groups)
   //-

   uint cnt13, cnt24;
   cnt13 = (lane1 >> 0u) & 0x00FF00FFu;
   cnt24 = (lane1 >> 8u) & 0x00FF00FFu;
   sum13 += cnt13;
   sum24 += cnt24;


   //-
   // Accumulate sums from 2nd lane (of 4 groups)
   //-

   cnt13 = (lane2 >> 0u) & 0x00FF00FFu;
   cnt24 = (lane2 >> 8u) & 0x00FF00FFu;
   sum13 += cnt13;
   sum24 += cnt24;


   //-
   // Accumulate sums from 3rd lane (of 4 groups)
   //-

   cnt13 = (lane3 >> 0u) & 0x00FF00FFu;
   cnt24 = (lane3 >> 8u) & 0x00FF00FFu;
   sum13 += cnt13;
   sum24 += cnt24;


   //-
   // Accumulate sums from 4th lane (of 4 groups)
   //-

   cnt13 = (lane4 >> 0u) & 0x00FF00FFu;
   cnt24 = (lane4 >> 8u) & 0x00FF00FFu;
   sum13 += cnt13;
   sum24 += cnt24;
}


/*-------------------------------------------------------------------
  Name:   AddThreadToRowCounts_V1
  Desc:   Accumulates 'Per Thread' counts into 'Per Row' Counts
 ------------------------------------------------------------------*/

template < uint BlockSize,     // Threads per Block
           uint BlockMask >    // Block Mask
__device__ __forceinline__
void AddThreadToRowCounts_V1
( 
   uint & rCnt1,    // OUT - 4 'per row' counts assigned to this thread
   uint & rCnt2,    //       ditto
   uint & rCnt3,    //       ditto
   uint & rCnt4,    //       ditto
   uint * basePtr,  // IN  - array of 'per thread' counts
   uint   tid
) 
{
   //-----
   // Serial Scan (Scan All 64 elements in sequence)
   //-----

   // Accumulate [0..63]
      // Note: Also zeros out [0..63]
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid +  0) );
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid +  4) );
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid +  8) );
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid + 12) );
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid + 16) );
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid + 20) );
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid + 24) );
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid + 28) );
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid + 32) );
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid + 36) );
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid + 40) );
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid + 44) );
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid + 48) );
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid + 52) );
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid + 56) );
   SS_Sums_4_Next_V1< BlockSize, BlockMask >( rCnt1, rCnt2, rCnt3, rCnt4, basePtr, (tid + 60) );
}


/*-------------------------------------------------------------------
  Name:   AddThreadToRowCounts_V2
  Desc:   Accumulates 'Per Thread' counts into 'Per Row' Counts
  Notes:   
  1. Vector Parallelism: 
       We accumulate 2 pairs at a time across each row 
       instead of 4 singletons for a big savings 
       in arithmetic operations.
  2. Overflow:
       We store 2 16-bit row sums per 32-bit number
       Which means that the accumulated Row sums need to not
       overflow a 16-bit number (65,535). 
       Since, we assume the maximum possible count per thread is 252
          64 threads * 252 =  16,128 <Safe>
         128 threads * 252 =  32,256 <Safe>
         256 threads * 252 =  64,512 <Safe>
         512 threads * 252 = 129,024 *** UNSAFE ***
       If this is a problem, revert to *_V1
  3. Register Pressure:
       *_V2 uses 6 more registers per thread than *_V1
       If this is a problem, revert to *_V1
 ------------------------------------------------------------------*/

template < uint BlockSize,     // Threads per Block
           uint BlockMask >    // BlockSize - 1
__device__ __forceinline__
void AddThreadToRowCounts_V2
( 
   uint & rCnt1,    // OUT - 4 'per row' counts assigned to this thread
   uint & rCnt2,    //       ditto
   uint & rCnt3,    //       ditto
   uint & rCnt4,    //       ditto
   uint * basePtr,  // IN  - array of 'per thread' counts
   uint   tid       // IN  - thread ID
) 
{
   uint sum13, sum24;
   sum13 = 0u;
   sum24 = 0u;

   //-----
   // Serial Scan (Scan All 64 elements in sequence)
   //-----

   // Accumulate Row Sums [0..63]
      // Note: Also zeros out count array while accumulating
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid +  0) );
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid +  4) );
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid +  8) );
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid + 12) );
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid + 16) );
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid + 20) );
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid + 24) );
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid + 28) );
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid + 32) );
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid + 36) );
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid + 40) );
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid + 44) );
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid + 48) );
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid + 52) );
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid + 56) );
   SS_Sums_4_Next_V2< BlockSize, BlockMask >( sum13, sum24, basePtr, (tid + 60) );

   // Convert row sums from pairs back into singletons
   uint sum1, sum2, sum3, sum4;
   sum1 = sum13 & 0x0000FFFFu;
   sum2 = sum24 & 0x0000FFFFu;
   sum3 = sum13 >> 16u;
   sum4 = sum24 >> 16u;

   // Add row sums back into register counts
   rCnt1 += sum1;
   rCnt2 += sum2;
   rCnt3 += sum3;
   rCnt4 += sum4;
}


/*---------------------------------------------------------
  Name:   H_K1_CountRows_256_TRISH
  Desc:   Bins elements into 256-way row counts
 ---------------------------------------------------------*/

template < uint logBankSize,   // log<2>( Channels per Bank )
           uint logWarpSize,	  // log<2>( Threads per Warp )
           uint BlockSize,     // Threads Per Block (needs to be a power of 2 & multiple of warpsize)
		   uint GridSize,	  // Blocks Per Grid
           uint K_length >     // #elements to process per thread before looping
__global__
void H_K1_CountRows_256_TRISH
( 
         uint * outRowCounts,  // OUT - 256-way row-sums array
   const uint * inVals,		   // IN  - values to bin and count
         uint   start,         // IN  - range [start,stop] to check and count
         uint   stop           //       ditto
) 
{
	//-------------------------------------------
	// Constant values (computed at compile time)
	//-------------------------------------------

		// Bank Size (elements per bank)
	const uint BankSize    = (1u << logBankSize);	   // 32 = 2^5 threads per bank
	const uint BankMask    = BankSize - 1u;	         // 31 = 32 - 1 = 0x1F = b11111
   const uint strideBank  = BankSize + 1u;          // 33 = 32 + 1
      // Extra '+1' to help try and avoid bank conflicts

		// Warp Size (threads per warp)
	const uint WarpSize    = (1u << logWarpSize);	   // 32 = 2^5 threads per warp
	const uint WarpMask    = WarpSize - 1u;			   // 31 = 32 - 1 = 0x1F = b11111

      // Block Size (threads per block)
   //const uint BlockSize   = 64u;
   const uint BlockMask   = BlockSize - 1u;

		// Chunk Size
	//const uint ChunkSize     = BlockSize * K_length;
   //const uint IN_WarpSize   = K_length * WarpSize;

      // K_length
   //const uint K_length = 16u;               // 16 
   const uint K4_length = K_length * 4u;      // 64 = 16 * 4
   const uint K4_stop   = 256u - K4_length;   // 192 = 256 - 64

		// Warps Per Block
	const uint WarpsPerBlock = BlockSize / WarpSize;   // 2 = 64/32

		// Bins per Histogram
	const uint nHistBins     = 256u;     // 256 = 2^8

		// Lane Info (Compress 4 'bins' into each 32-bit value)
	const uint nLanes		   = 64u;   // 64, # Lanes = 256 bins / 4 bins per lane

		// 'Per Thread' counts array
	const uint nTCounts      = nLanes * BlockSize;
	const uint banksTCounts  = (nTCounts + BankMask) / BankSize;
	const uint padTCounts    = (banksTCounts * BankSize) - nTCounts;
	const uint sizeTCounts   = nTCounts + padTCounts;

      // Output size
   const uint OutWarpSize   = nHistBins / WarpsPerBlock;
   const uint OutLength     = OutWarpSize / WarpSize;
   const uint OutStrideSize = OutLength * strideBank;

		// Array Initialization
	const uint nPassesThrd  = sizeTCounts / BlockSize;
	const uint leftOverThrd = sizeTCounts - (nPassesThrd * BlockSize);

	const uint nThreadsPerGrid = BlockSize * GridSize;	//   3,072 = 64 * 48
   const uint rowSize = K_length * nThreadsPerGrid;		// 193,586 = 63 * 64 * 48


	//------------------------------------
	// Local Variables
	//------------------------------------

		// Local variables (shared memory)
	__shared__ uint  s_thrdCounts[sizeTCounts];   // 'per thread' counts

      // Local variables (registers)
   uint rowCnt1 = 0u;
   uint rowCnt2 = 0u;
   uint rowCnt3 = 0u; 
   uint rowCnt4 = 0u;

	//---------------------------
	// Compute Indices & Pointers
	//---------------------------

   uint tid = threadIdx.x;		// Thread ID within Block
   uint * cntPtr;
   uint * basePtr;

   {
      // Get Warp Row & Column
      //uint warpRow = threadIdx.x >> logWarpSize; // tid / 32
      //uint warpCol = threadIdx.x & WarpMask;     // tid % 32

      // Compute starting 'input' offset (Warp Sequential Layout)
      //inIdx = (warpRow * IN_WarpSize) // Move to each warps assigned portion of work
      //        + warpCol;              // Move to warp column (in warp)

         // Compute starting serial scan index
      uint baseIdx = (tid * BlockSize);

         // Get pointers into shared memory array
         // for different views of memory
      cntPtr  = &s_thrdCounts[threadIdx.x];
      basePtr = &s_thrdCounts[baseIdx];
   }


	//-------------------------------------------
	// Zero out arrays
	//-------------------------------------------

   {
	   //-
	   // Zero out 'Per Thread' counts
	   //-

      uint * ptrTC = (&s_thrdCounts[0]);
      SetArray_BlockSeq
         < 
            uint, BlockSize, nPassesThrd, leftOverThrd, sizeTCounts
         >
         ( 
            ptrTC, 0u
         );
   }

	// Sync Threads in Block
	if (WarpsPerBlock >= 2u) { __syncthreads(); }

   //-----
   // Compute thread, block, & grid indices & sizes
   //-----
 
   uint bid = (blockIdx.y * gridDim.x) + blockIdx.x;		// Block ID within Grid
   uint elemOffset = (bid * K_length * BlockSize) + tid;	// Starting offset 

   uint nElems32        = stop - start + 1u;
   uint nMaxRows        = (nElems32 + (rowSize - 1u)) / rowSize;
   uint nSafeRows       = nElems32 / rowSize;
   uint nSafeElems      = nSafeRows * rowSize;
   uint nLeftOverElems  = nElems32 - nSafeElems;

   uint startIdx        = start + elemOffset;
   uint stopIdx         = startIdx + (nSafeRows * rowSize);
   uint currIdx         = startIdx;
   uint overflow        = 0u;


   //-----
   // Process all safe blocks
   //-----

   // 'input' pointer for reading from memory
   const uint * inPtr = &inVals[currIdx];

   while (currIdx < stopIdx)
	{
      //-----
      // Accumulate 'thread' counts into 'row' counts
      //    Note: Also zeros out 'per thread' count array
      //-----

      if (overflow >= K4_stop)
      {
         // Sync Threads in Block
         if (WarpsPerBlock >= 2u) { __syncthreads(); }

         //AddThreadToRowCounts_V1< BlockSize, BlockMask >( rowCnt1, rowCnt2, rowCnt3, rowCnt4, basePtr, tid );
         AddThreadToRowCounts_V2< BlockSize, BlockMask >( rowCnt1, rowCnt2, rowCnt3, rowCnt4, basePtr, tid );
         overflow = 0u;

         // Sync Threads in Block
         if (WarpsPerBlock >= 2u) { __syncthreads(); }
      }

      uint val1, val2, val3, val4;

         // NOTE:  the 'K_length' variable below is a static
         //        hard-coded constant in the range [1..63].
         //        K = 'Work per thread' per loop (stride)...
         //        The compiler will take care of throwing away 
         //        any unused code greater than our specified 'K'
         //        value, with no negative impact on performance.

      //-
      // Process values [0..3] (bytes 0..15)
      //-

      // Read in first 'four' values (32-bit)
      if (K_length >= 1u) { val1 = inPtr[0u*BlockSize]; }
      if (K_length >= 2u) { val2 = inPtr[1u*BlockSize]; }
      if (K_length >= 3u) { val3 = inPtr[2u*BlockSize]; }
      if (K_length >= 4u) { val4 = inPtr[3u*BlockSize]; }

      // Bin first 'four' values
      if (K_length >= 1u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 2u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 3u) { Bin4_None<BlockSize>( cntPtr, val3 ); }
      if (K_length >= 4u) { Bin4_None<BlockSize>( cntPtr, val4 ); }


      //-
      // Process values [4..7] (bytes 16..31)
      //-

      // Read in next 'four' values (32-bit)
      if (K_length >= 5u) { val1 = inPtr[4u*BlockSize]; }
      if (K_length >= 6u) { val2 = inPtr[5u*BlockSize]; }
      if (K_length >= 7u) { val3 = inPtr[6u*BlockSize]; }
      if (K_length >= 8u) { val4 = inPtr[7u*BlockSize]; }

      // Bin 'four' values (4 bytes at a time)
      if (K_length >= 5u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 6u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 7u) { Bin4_None<BlockSize>( cntPtr, val3 ); }
      if (K_length >= 8u) { Bin4_None<BlockSize>( cntPtr, val4 ); }


      //-
      // Process values [8..11] (bytes 32..47)
      //-

      // Read in next 'four' values (32-bit)
      if (K_length >=  9u) { val1 = inPtr[ 8u*BlockSize]; } 
      if (K_length >= 10u) { val2 = inPtr[ 9u*BlockSize]; }
      if (K_length >= 11u) { val3 = inPtr[10u*BlockSize]; }
      if (K_length >= 12u) { val4 = inPtr[11u*BlockSize]; }

      // Bin 'four' values (4 bytes at a time)
      if (K_length >=  9u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 10u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 11u) { Bin4_None<BlockSize>( cntPtr, val3 ); }
      if (K_length >= 12u) { Bin4_None<BlockSize>( cntPtr, val4 ); }

      //-
      // Process values [12..15] (bytes 48..63)
      //-

      // Read in next 'four' values (32-bit)
      if (K_length >= 13u) { val1 = inPtr[12u*BlockSize]; }
      if (K_length >= 14u) { val2 = inPtr[13u*BlockSize]; }
      if (K_length >= 15u) { val3 = inPtr[14u*BlockSize]; }
      if (K_length >= 16u) { val4 = inPtr[15u*BlockSize]; }

      // Bin 'four' values (4 bytes at a time)
      if (K_length >= 13u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 14u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 15u) { Bin4_None<BlockSize>( cntPtr, val3 ); }
      if (K_length >= 16u) { Bin4_None<BlockSize>( cntPtr, val4 ); }


      //-
      // Process values [16..19] (bytes 64..79)
      //-

      // Read in next 'four' values (32-bit)
      if (K_length >= 17u) { val1 = inPtr[16u*BlockSize]; }
      if (K_length >= 18u) { val2 = inPtr[17u*BlockSize]; }
      if (K_length >= 19u) { val3 = inPtr[18u*BlockSize]; }
      if (K_length >= 20u) { val4 = inPtr[19u*BlockSize]; }

      // Bin 'four' values (4 bytes at a time)
      if (K_length >= 17u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 18u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 19u) { Bin4_None<BlockSize>( cntPtr, val3 ); }
      if (K_length >= 20u) { Bin4_None<BlockSize>( cntPtr, val4 ); }


      //-
      // Process values [20..23] (bytes 80..95)
      //-

      // Read in next 'four' values (32-bit)
      if (K_length >= 21u) { val1 = inPtr[20u*BlockSize]; }
      if (K_length >= 22u) { val2 = inPtr[21u*BlockSize]; }
      if (K_length >= 23u) { val3 = inPtr[22u*BlockSize]; }
      if (K_length >= 24u) { val4 = inPtr[23u*BlockSize]; }

      // Bin 'four' values (4 bytes at a time)
      if (K_length >= 21u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 22u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 23u) { Bin4_None<BlockSize>( cntPtr, val3 ); }
      if (K_length >= 24u) { Bin4_None<BlockSize>( cntPtr, val4 ); }


      //-
      // Process values [24..27] (bytes 96..111)
      //-

      // Read in next 'four' values (32-bit)
      if (K_length >= 25u) { val1 = inPtr[24u*BlockSize]; }
      if (K_length >= 26u) { val2 = inPtr[25u*BlockSize]; }
      if (K_length >= 27u) { val3 = inPtr[26u*BlockSize]; }
      if (K_length >= 28u) { val4 = inPtr[27u*BlockSize]; }

      // Bin 'four' values (4 bytes at a time)
      if (K_length >= 25u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 26u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 27u) { Bin4_None<BlockSize>( cntPtr, val3 ); }
      if (K_length >= 28u) { Bin4_None<BlockSize>( cntPtr, val4 ); }


      //-
      // Process values [28..31] (bytes 112..127)
      //-

      // Read in next 'four' values (32-bit)
      if (K_length >= 29u) { val1 = inPtr[28u*BlockSize]; }
      if (K_length >= 30u) { val2 = inPtr[29u*BlockSize]; }
      if (K_length >= 31u) { val3 = inPtr[30u*BlockSize]; }
      if (K_length >= 32u) { val4 = inPtr[31u*BlockSize]; }

      // Bin 'four' values (4 bytes at a time)
      if (K_length >= 29u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 30u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 31u) { Bin4_None<BlockSize>( cntPtr, val3 ); }
      if (K_length >= 32u) { Bin4_None<BlockSize>( cntPtr, val4 ); }


      //-
      // Process values [32..35] (bytes 128..143)
      //-

      // Read in next 'four' values (32-bit)
      if (K_length >= 33u) { val1 = inPtr[32u*BlockSize]; }
      if (K_length >= 34u) { val2 = inPtr[33u*BlockSize]; }
      if (K_length >= 35u) { val3 = inPtr[34u*BlockSize]; }
      if (K_length >= 36u) { val4 = inPtr[35u*BlockSize]; }

      // Bin 'four' values (4 bytes at a time)
      if (K_length >= 33u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 34u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 35u) { Bin4_None<BlockSize>( cntPtr, val3 ); }
      if (K_length >= 36u) { Bin4_None<BlockSize>( cntPtr, val4 ); }


      //-
      // Process values [36..39] (bytes 144..159)
      //-

      // Read in next 'four' values (32-bit)
      if (K_length >= 37u) { val1 = inPtr[36u*BlockSize]; }
      if (K_length >= 38u) { val2 = inPtr[37u*BlockSize]; }
      if (K_length >= 39u) { val3 = inPtr[38u*BlockSize]; }
      if (K_length >= 40u) { val4 = inPtr[39u*BlockSize]; }

      // Bin 'four' values (4 bytes at a time)
      if (K_length >= 37u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 38u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 39u) { Bin4_None<BlockSize>( cntPtr, val3 ); }
      if (K_length >= 40u) { Bin4_None<BlockSize>( cntPtr, val4 ); }

      //-
      // Process values [40..43] (bytes 160-175)
      //-

      // Read in next 'four' values (32-bit)
      if (K_length >= 41u) { val1 = inPtr[40u*BlockSize]; }
      if (K_length >= 42u) { val2 = inPtr[41u*BlockSize]; }
      if (K_length >= 43u) { val3 = inPtr[42u*BlockSize]; }
      if (K_length >= 44u) { val4 = inPtr[43u*BlockSize]; }

      // Bin 'four' values (4 bytes at a time)
      if (K_length >= 41u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 42u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 43u) { Bin4_None<BlockSize>( cntPtr, val3 ); }
      if (K_length >= 44u) { Bin4_None<BlockSize>( cntPtr, val4 ); }


      //-
      // Process values [44..47] (bytes 176-191)
      //-

      // Read in next 'four' values (32-bit)
      if (K_length >= 45u) { val1 = inPtr[44u*BlockSize]; }
      if (K_length >= 46u) { val2 = inPtr[45u*BlockSize]; }
      if (K_length >= 47u) { val3 = inPtr[46u*BlockSize]; }
      if (K_length >= 48u) { val4 = inPtr[47u*BlockSize]; }

      // Bin 'four' values (4 bytes at a time)
      if (K_length >= 45u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 46u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 47u) { Bin4_None<BlockSize>( cntPtr, val3 ); }
      if (K_length >= 48u) { Bin4_None<BlockSize>( cntPtr, val4 ); }


      //-
      // Process values [48-51] (bytes 192-207)
      //-

      // Read in next 'four' values (32-bit)
      if (K_length >= 49u) { val1 = inPtr[48u*BlockSize]; }
      if (K_length >= 50u) { val2 = inPtr[49u*BlockSize]; }
      if (K_length >= 51u) { val3 = inPtr[50u*BlockSize]; }
      if (K_length >= 52u) { val4 = inPtr[51u*BlockSize]; }

      // Bin 'four' values (4 bytes at a time)
      if (K_length >= 49u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 50u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 51u) { Bin4_None<BlockSize>( cntPtr, val3 ); }
      if (K_length >= 52u) { Bin4_None<BlockSize>( cntPtr, val4 ); }


      //-
      // Process values [52-55] (bytes 208-223)
      //-

      // Read in next 'four' values (32-bit)
      if (K_length >= 53u) { val1 = inPtr[52u*BlockSize]; }
      if (K_length >= 54u) { val2 = inPtr[53u*BlockSize]; }
      if (K_length >= 55u) { val3 = inPtr[54u*BlockSize]; }
      if (K_length >= 56u) { val4 = inPtr[55u*BlockSize]; }

      // Bin 'four' values (4 bytes at a time)
      if (K_length >= 53u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 54u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 55u) { Bin4_None<BlockSize>( cntPtr, val3 ); }
      if (K_length >= 56u) { Bin4_None<BlockSize>( cntPtr, val4 ); }


      //-
      // Process values [56-59] (bytes 224-239)
      //-

      // Read in next 'four' values (32-bit)
      if (K_length >= 57u) { val1 = inPtr[56u*BlockSize]; }
      if (K_length >= 58u) { val2 = inPtr[57u*BlockSize]; }
      if (K_length >= 59u) { val3 = inPtr[58u*BlockSize]; }
      if (K_length >= 60u) { val4 = inPtr[59u*BlockSize]; }

      // Bin 'four' values (4 bytes at a time)
      if (K_length >= 57u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 58u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 59u) { Bin4_None<BlockSize>( cntPtr, val3 ); }
      if (K_length >= 60u) { Bin4_None<BlockSize>( cntPtr, val4 ); }


      //-
      // Process values [60-62] (bytes 240-251)
      //-
         // Note: We deliberately do not support k >= '64' to
         //       avoid overflow issues during 'binning'
         //       As our 'per thread' 'bin counts' can only handle 
         //       '255' increments before overflow becomes a problem.
         //       and 252 is the next smallest number 
         //       evenly divisible by 4, IE 4 bytes per 32-bit value
         //       63 values = 252 bytes / 4 bytes per value.

      // Read in next 'four' values (32-bit)
      if (K_length >= 61u) { val1 = inPtr[60u*BlockSize]; }
      if (K_length >= 62u) { val2 = inPtr[61u*BlockSize]; }
      if (K_length >= 63u) { val3 = inPtr[62u*BlockSize]; }

      // Note: Do not uncomment => *OVERFLOW* bug !!!
      //if (K_length >= 64u) { val4 = inPtr[63u*BlockSize]; }

      // Bin 'four' values (4 bytes at a time)
      if (K_length >= 61u) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      if (K_length >= 62u) { Bin4_None<BlockSize>( cntPtr, val2 ); }
      if (K_length >= 63u) { Bin4_None<BlockSize>( cntPtr, val3 ); }

      // Note: Do not uncomment => *OVERFLOW* bug !!!
      //if (K_length >= 64u) { Bin4_None<BlockSize>( cntPtr, val4 ); }


      // Increment 'overflow' count
      overflow += K4_length;   // K values * 4 bytes per value


      //-----
		// Move to next row of work
		//-----

		currIdx += rowSize;
        inPtr += rowSize;
	}


	//--------------------------------------
	// LAST: Process last leftover chunk
    //       with more careful range checking
	//--------------------------------------

	if (nLeftOverElems)
	{
      //-----
      // Accumulate 'thread' counts into 'row' counts
      //    Note: Also zeros out 'per thread' count array
      //-----

      if (overflow >= K4_stop)
      {
         // Sync Threads in Block
         if (WarpsPerBlock >= 2u) { __syncthreads(); }

         //AddThreadToRowCounts_V1< BlockSize, BlockMask >( rowCnt1, rowCnt2, rowCnt3, rowCnt4, basePtr, tid );
         AddThreadToRowCounts_V2< BlockSize, BlockMask >( rowCnt1, rowCnt2, rowCnt3, rowCnt4, basePtr, tid );
         overflow = 0u;

         // Sync Threads in Block
         if (WarpsPerBlock >= 2u) { __syncthreads(); }
      }

         // NOTE #1:  the 'K_length' variable below is a static
         //           hard-coded constant in the range [1..63].
         //           K = 'Work per thread' per loop (stride)...
         //           The compiler will take care of throwing away 
         //           any unused code greater than our specified 'K'
         //           value, with no negative impact on performance.

         // NOTE #2:  We use a cooperative stride 
         //           across each thread in each block in grid
         //           ChunkSize = BlockSize * GridSize = 64 * 48 = 3072
         //           RowSize   = WorkPerThead(K) * ChunkSize = 63 * 3072 = 193,536
         // 
         //                       B0   B1  ...  B47  (Blocks in Grid)
         //                      ---- ---- --- ----
         //           k =  1 =>  |64| |64| ... |64|  (3072 Thread & I/O requests for 1st work item per thread)
         //           k =  2 =>  |64| |64| ... |64|  ditto (2nd work item per thread)
         //               ...       ...         ...
         //           k = 63 =>  |64| |64| ... |64|  ditto (63 work item per thread)

         // NOTE #3:  We use "Divide & Conquer" to avoid as much slower range checking as possible
         //			  Try batches of 32, 16, 8, 4, 2, 1, and finally leftover (on which we finally must range check) 

      //----
      // Setup Pointers & Indices for cooperative stride 
      //----

      uint bid        = (blockIdx.y * gridDim.x) + blockIdx.x;	// Get block index
      uint nSkip      = nSafeRows * rowSize;						// Skip past already processed rows
      uint chunkIdx   = (bid * BlockSize) + tid;					// Get starting index within chunk
      uint baseIdx    = start + nSkip + chunkIdx;				// Get starting index for left over elements

      uint val1, val2, val3, val4;

      //------
      // Try Section of 32
      //------

      if (K_length >= 32u)
      {
         // Process 32 chunks safely without range checking
         if (nLeftOverElems >= (32u * nThreadsPerGrid))
         {
            // Get pointer
            inPtr = &inVals[baseIdx];

            //-----
            // Read & Bin [1..4]
            //-----

            val1 = inPtr[(0u*nThreadsPerGrid)];
            val2 = inPtr[(1u*nThreadsPerGrid)];
            val3 = inPtr[(2u*nThreadsPerGrid)];
            val4 = inPtr[(3u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );
            Bin4_None<BlockSize>( cntPtr, val3 );
            Bin4_None<BlockSize>( cntPtr, val4 );


            //-----
            // Read & Bin [5..8]
            //-----

            val1 = inPtr[(4u*nThreadsPerGrid)];
            val2 = inPtr[(5u*nThreadsPerGrid)];
            val3 = inPtr[(6u*nThreadsPerGrid)];
            val4 = inPtr[(7u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );
            Bin4_None<BlockSize>( cntPtr, val3 );
            Bin4_None<BlockSize>( cntPtr, val4 );


            //-----
            // Read & Bin [9..12]
            //-----

            val1 = inPtr[( 8u*nThreadsPerGrid)];
            val2 = inPtr[( 9u*nThreadsPerGrid)];
            val3 = inPtr[(10u*nThreadsPerGrid)];
            val4 = inPtr[(11u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );
            Bin4_None<BlockSize>( cntPtr, val3 );
            Bin4_None<BlockSize>( cntPtr, val4 );


            //-----
            // Read & Bin [13..16]
            //-----

            val1 = inPtr[(12u*nThreadsPerGrid)];
            val2 = inPtr[(13u*nThreadsPerGrid)];
            val3 = inPtr[(14u*nThreadsPerGrid)];
            val4 = inPtr[(15u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );
            Bin4_None<BlockSize>( cntPtr, val3 );
            Bin4_None<BlockSize>( cntPtr, val4 );


            //-----
            // Read & Bin [17..20]
            //-----

            val1 = inPtr[(16u*nThreadsPerGrid)];
            val2 = inPtr[(17u*nThreadsPerGrid)];
            val3 = inPtr[(18u*nThreadsPerGrid)];
            val4 = inPtr[(19u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );
            Bin4_None<BlockSize>( cntPtr, val3 );
            Bin4_None<BlockSize>( cntPtr, val4 );


            //-----
            // Read & Bin [21..24]
            //-----

            val1 = inPtr[(20u*nThreadsPerGrid)];
            val2 = inPtr[(21u*nThreadsPerGrid)];
            val3 = inPtr[(22u*nThreadsPerGrid)];
            val4 = inPtr[(23u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );
            Bin4_None<BlockSize>( cntPtr, val3 );
            Bin4_None<BlockSize>( cntPtr, val4 );


            //-----
            // Read & Bin [25..28]
            //-----

            val1 = inPtr[(24u*nThreadsPerGrid)];
            val2 = inPtr[(25u*nThreadsPerGrid)];
            val3 = inPtr[(26u*nThreadsPerGrid)];
            val4 = inPtr[(27u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );
            Bin4_None<BlockSize>( cntPtr, val3 );
            Bin4_None<BlockSize>( cntPtr, val4 );


            //-----
            // Read & Bin [29..32]
            //-----

            val1 = inPtr[(28u*nThreadsPerGrid)];
            val2 = inPtr[(29u*nThreadsPerGrid)];
            val3 = inPtr[(30u*nThreadsPerGrid)];
            val4 = inPtr[(31u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );
            Bin4_None<BlockSize>( cntPtr, val3 );
            Bin4_None<BlockSize>( cntPtr, val4 );


            // Move to next section
            baseIdx        += (32u * nThreadsPerGrid);
            nLeftOverElems -= (32u * nThreadsPerGrid);
         }
      }


      //------
      // Try Section of 16
      //------

      if (K_length >= 16u)
      {
         // Process 16 chunks safely without range checking
         if (nLeftOverElems >= (16u * nThreadsPerGrid))
         {
            // Get pointer
            inPtr = &inVals[baseIdx];

            //-----
            // Read & Bin [1..4]
            //-----

            val1 = inPtr[(0u*nThreadsPerGrid)];
            val2 = inPtr[(1u*nThreadsPerGrid)];
            val3 = inPtr[(2u*nThreadsPerGrid)];
            val4 = inPtr[(3u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );
            Bin4_None<BlockSize>( cntPtr, val3 );
            Bin4_None<BlockSize>( cntPtr, val4 );


            //-----
            // Read & Bin [5..8]
            //-----

            val1 = inPtr[(4u*nThreadsPerGrid)];
            val2 = inPtr[(5u*nThreadsPerGrid)];
            val3 = inPtr[(6u*nThreadsPerGrid)];
            val4 = inPtr[(7u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );
            Bin4_None<BlockSize>( cntPtr, val3 );
            Bin4_None<BlockSize>( cntPtr, val4 );


            //-----
            // Read & Bin [9..12]
            //-----

            val1 = inPtr[( 8u*nThreadsPerGrid)];
            val2 = inPtr[( 9u*nThreadsPerGrid)];
            val3 = inPtr[(10u*nThreadsPerGrid)];
            val4 = inPtr[(11u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );
            Bin4_None<BlockSize>( cntPtr, val3 );
            Bin4_None<BlockSize>( cntPtr, val4 );


            //-----
            // Read & Bin [13..16]
            //-----

            val1 = inPtr[(12u*nThreadsPerGrid)];
            val2 = inPtr[(13u*nThreadsPerGrid)];
            val3 = inPtr[(14u*nThreadsPerGrid)];
            val4 = inPtr[(15u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );
            Bin4_None<BlockSize>( cntPtr, val3 );
            Bin4_None<BlockSize>( cntPtr, val4 );


            // Move to next section
            baseIdx        += (16u * nThreadsPerGrid);
            nLeftOverElems -= (16u * nThreadsPerGrid);
         }
      }


      //------
      // Try Section of 8
      //------

      if (K_length >= 8u)
      {
         // Process 8 chunks safely without range checking
         if (nLeftOverElems >= (8u * nThreadsPerGrid))
         {
            // Get pointer
            inPtr = &inVals[baseIdx];

            //-----
            // Read & Bin [1..4]
            //-----

            val1 = inPtr[(0u*nThreadsPerGrid)];
            val2 = inPtr[(1u*nThreadsPerGrid)];
            val3 = inPtr[(2u*nThreadsPerGrid)];
            val4 = inPtr[(3u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );
            Bin4_None<BlockSize>( cntPtr, val3 );
            Bin4_None<BlockSize>( cntPtr, val4 );


            //-----
            // Read & Bin [5..8]
            //-----

            val1 = inPtr[(4u*nThreadsPerGrid)];
            val2 = inPtr[(5u*nThreadsPerGrid)];
            val3 = inPtr[(6u*nThreadsPerGrid)];
            val4 = inPtr[(7u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );
            Bin4_None<BlockSize>( cntPtr, val3 );
            Bin4_None<BlockSize>( cntPtr, val4 );


            // Move to next section
            baseIdx        += (8u * nThreadsPerGrid);
            nLeftOverElems -= (8u * nThreadsPerGrid);
         }
      }


      //------
      // Try Section of 4
      //------

      if (K_length >= 4u)
      {
         // Process 4 chunks safely without range checking
         if (nLeftOverElems >= (4u * nThreadsPerGrid))
         {
            // Get pointer
            inPtr = &inVals[baseIdx];

            //-----
            // Read & Bin [1..4]
            //-----

            val1 = inPtr[(0u*nThreadsPerGrid)];
            val2 = inPtr[(1u*nThreadsPerGrid)];
            val3 = inPtr[(2u*nThreadsPerGrid)];
            val4 = inPtr[(3u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );
            Bin4_None<BlockSize>( cntPtr, val3 );
            Bin4_None<BlockSize>( cntPtr, val4 );


            // Move to next section
            baseIdx        += (4u * nThreadsPerGrid);
            nLeftOverElems -= (4u * nThreadsPerGrid);
         }
      }


      //------
      // Try Section of 2
      //------

      if (K_length >= 2u)
      {
         // Process 2 chunks safely without range checking
         if (nLeftOverElems >= (2u * nThreadsPerGrid))
         {
            // Get pointer
            inPtr = &inVals[baseIdx];

            //-----
            // Read & Bin [1..2]
            //-----

            val1 = inPtr[(0u*nThreadsPerGrid)];
            val2 = inPtr[(1u*nThreadsPerGrid)];

            Bin4_None<BlockSize>( cntPtr, val1 );
            Bin4_None<BlockSize>( cntPtr, val2 );


            // Move to next section
            baseIdx        += (2u * nThreadsPerGrid);
            nLeftOverElems -= (2u * nThreadsPerGrid);
         }
      }


      //------
      // Try Section of 1
      //------

      if (K_length >= 1u)
      {
         // Process 1 chunk safely without range checking
         if (nLeftOverElems >= (1u * nThreadsPerGrid))
         {
            // Get pointer
            inPtr = &inVals[baseIdx];

            //-----
            // Read & Bin [1]
            //-----

            val1 = inPtr[(0u*nThreadsPerGrid)];
            Bin4_None<BlockSize>( cntPtr, val1 );

            // Move to next section
            baseIdx        += (1u * nThreadsPerGrid);
            nLeftOverElems -= (1u * nThreadsPerGrid);
         }
      }


      //------
      // Process Last few elements
      //    while carefully RANGE CHECKING !!!
      //------

      if (nLeftOverElems > 0u)
      {
         // Make sure we are 'in range' before reading
         uint inRange1 = (baseIdx <= stop);

         // Read in 32-bit element, only if *safely* in range
         if (inRange1) { val1 = inVals[baseIdx]; }

         // Bin 'four' values in 32-bit element (4 bytes at a time)
         if (inRange1) { Bin4_None<BlockSize>( cntPtr, val1 ); }
      }

      // Update Accumulation count
      overflow += K4_length;   // 64 = 16 elems * 4 bytes per elem
	}


   //-----
   // Accumulate 'thread' counts into 'row' counts
   //    Note: Also zeros out 'per thread' count array
   //-----

   if (overflow > 0u)
   {
      // Sync Threads in Block
      if (WarpsPerBlock >= 2u) { __syncthreads(); }

      //AddThreadToRowCounts_V1< BlockSize, BlockMask >( rowCnt1, rowCnt2, rowCnt3, rowCnt4, basePtr, tid );
      AddThreadToRowCounts_V2< BlockSize, BlockMask >( rowCnt1, rowCnt2, rowCnt3, rowCnt4, basePtr, tid );
      overflow = 0u;

      // Sync Threads in Block
      if (WarpsPerBlock >= 2u) { __syncthreads(); }
   }


	//-------------------------------------------------
	// Write out final row 'counts'
	//-------------------------------------------------

   {
      // Compute starting 'row counts' offset
      uint rIdx = threadIdx.x * 4u;         // 4 groups per lane
      uint rRow = rIdx >> logBankSize;
      uint rCol = rIdx & BankMask;

      uint rowIdx = (rRow * strideBank) + (rCol + 1u);
         // Extra '+1' to shift past initial pad element      

      uint * rowPtr = &s_thrdCounts[rowIdx];

      // Store row counts in row array
      rowPtr[0] = rowCnt1;
      rowPtr[1] = rowCnt2;
      rowPtr[2] = rowCnt3;
      rowPtr[3] = rowCnt4;

      // Sync Threads in Block
      if (WarpsPerBlock >= 2u) { __syncthreads(); }

      // Get Warp Row & Column
      uint warpRow = threadIdx.x >> logWarpSize; // tid / 32
      uint warpCol = threadIdx.x & WarpMask;     // tid % 32

      // Get local & global indices
      uint outGlobal = (blockIdx.x * nHistBins);
      uint outLocal  = (warpRow * OutWarpSize);
      uint rowBase   = (warpRow * OutStrideSize);
      uint outBase   = outGlobal + outLocal;
      uint rowOff    = warpCol + 1u;

      uint outIdx = outBase + warpCol;
          rowIdx = rowBase + rowOff;

      // Get local & global pointers
      uint * outPtr = &outRowCounts[outIdx];
            rowPtr = &s_thrdCounts[rowIdx];

         // Write our 'per row' counts in warp sequential order
      if (OutLength >= 1u) { outPtr[(0u*WarpSize)] = rowPtr[(0u*strideBank)]; }
      if (OutLength >= 2u) { outPtr[(1u*WarpSize)] = rowPtr[(1u*strideBank)]; }
      if (OutLength >= 3u) { outPtr[(2u*WarpSize)] = rowPtr[(2u*strideBank)]; }
      if (OutLength >= 4u) { outPtr[(3u*WarpSize)] = rowPtr[(3u*strideBank)]; }
      if (OutLength >= 5u) { outPtr[(4u*WarpSize)] = rowPtr[(4u*strideBank)]; }
      if (OutLength >= 6u) { outPtr[(5u*WarpSize)] = rowPtr[(5u*strideBank)]; }
      if (OutLength >= 7u) { outPtr[(6u*WarpSize)] = rowPtr[(6u*strideBank)]; }
      if (OutLength >= 8u) { outPtr[(7u*WarpSize)] = rowPtr[(7u*strideBank)]; }
   }
}


//-----------------------------------------------
// Name: H_K2_RowCounts_To_RowStarts_256
// Desc: Sum 256-way 'per row' counts into 
//       total 256-way counts using prefix-sum
//------------------------------------------------

template < uint logBankSize,		// log<2>( Channels per Bank )
           uint logWarpSize,		// log<2>( Threads Per Warp )
           uint BlockSize >	      // Threads Per Block
__global__
void H_K2_RowCounts_To_RowStarts_256
( 
         uint * outTotalCounts,	// OUT - total counts
         uint * outTotalStarts,	// OUT - total starts
         uint * outRowStarts,	   // OUT - row starts
	const uint * inRowCounts,	   // IN  - 'per row' counts to accumulate
         uint   nRows			      // IN  - number of rows to accumulate
) 
{
	//------------------------------------
	// Constant values
	//------------------------------------

		// Memory Channels Per Bank
	const uint BankSize  = 1u << logBankSize;	// 32 (or 16)
	const uint BankMask  = BankSize - 1u;	   // 31 (or 15)

		// Threads Per Warp
	const uint WarpSize  = 1u << logWarpSize;	// 32
	const uint WarpMask  = WarpSize - 1u;      // 31

		// Warps Per Block
	const uint WarpsPerBlock = BlockSize / WarpSize; // 8 = 256 / 32
	
		// Size of 'Row Counts' and 'Row Starts' array
	//const uint nElemsCounts = 256;
	//const uint banksCounts  = (nElemsCounts + BankMask) / BankSize;
	//const uint padCounts    = ((banksCounts * BankSize) - nElemsCounts);
	//const uint sizeCounts   = nElemsCounts + padCounts;

      // Stride for padded bank of elements
   const uint strideBank = 1u + BankSize;

		// Serial Scan Array
   const uint nSS1      = 256u + 2u;
   const uint nRowsSS1  = (nSS1 + BankMask) / BankSize;
	const uint nElemsSS1 = nRowsSS1 * strideBank;
	const uint banksSS1  = (nElemsSS1 + BankMask) / BankSize;
	const uint padSS1    = ((banksSS1 * BankSize) - nElemsSS1);
	const uint sizeSS1   = nElemsSS1 + padSS1;

		// WarpScan array
	const uint strideWS2 = WarpSize
		                   + (WarpSize >> 1u)
						       + 1u;			// 49 = (32 + 16 + 1)
   const uint nWarpsWS2 = 1u;
	const uint nElemsWS2 = nWarpsWS2 * strideWS2;
	const uint banksWS2  = (nElemsWS2 + BankMask) / BankSize;
	const uint padWS2    = ((banksWS2 * BankSize) - nElemsWS2);
	const uint sizeWS2   = nElemsWS2 + padWS2;

	//const uint nSafePassesCnts = sizeCounts / BlockSize;
	//const uint leftOverCnts    = sizeCounts - (nSafePassesCnts * BlockSize);

	const uint nSafePassesSS1  = sizeSS1 / BlockSize;
	const uint leftOverSS1     = sizeSS1 - (nSafePassesSS1 * BlockSize);

	const uint nSafePassesWS2  = sizeWS2 / BlockSize;
	const uint leftOverWS2     = sizeWS2 - (nSafePassesWS2 * BlockSize);


	//------------------------------------
	// Local variables
	//------------------------------------

		// shared memory
	//__shared__ uint s_rowStarts[sizeCounts];	// 'Row Starts' one chunk at a time
   __shared__ uint s_ss1[sizeSS1];            // Used for serial scan
	__shared__ uint s_ws2[sizeWS2];		      // Used for parallel warp scan

		// Registers
	uint tSum;				// Per thread accumulator

	//------------------------------------
	// Compute Indices & Pointers
	//------------------------------------

   uint warpRow, warpCol;
   uint storeIdx, prevIdx, ss1Idx, ws2Idx;
   {
      // Compute Bank Offsets
	   //uint bankRow = threadIdx.x >> logBankSize;		// tid / 32
	   uint bankCol = threadIdx.x & BankMask;			// tid % 32

	   // Compute warp offsets
	   warpRow = threadIdx.x >> logWarpSize;		// tid / 32
	   warpCol = threadIdx.x & WarpMask;			// tid % 32

      // Compute Store index (for storing final counts before prefix sum)
      uint sIdx = threadIdx.x;
      uint storeRow = sIdx >> logBankSize;   // tid / 32
      uint storeCol = sIdx & BankMask;       // tid % 32
      storeIdx = (storeRow * strideBank)
                 + storeCol
                 + 2u;        // Pad for 'reach back'

	      //--
	      // Previous Column (Serial Scan 1)
	      //   1.) Reach back one column
	      //   2.) But, we need to skip over extra padding before the first
         //       thread in every bank, so reach back two columns
         // However, the very first thread in the very first bank needs
         // to be able to reach back safely 2 columns without going 'out of range'.
         //
         // We work around this by pre-padding the 's_ss1' array with
         // an extra 2 elements and shifting indices over by two as needed to skip over padding.
	      //--

 	   uint prevCol = ((bankCol == 0u) ? 2u : 1u);
      prevIdx = storeIdx - prevCol;

      // Compute Serial Scan index
      uint ssIdx  = threadIdx.x * 8u;
      uint ss1Row = ssIdx >> logBankSize;   // (tid*8) / 32
      uint ss1Col = ssIdx & BankMask;       // (tid*8) % 32
      ss1Idx = (ss1Row * strideBank)
               + ss1Col
               + 2u;       // pad for 'reach back'

	   // Compute Warp Scan Index
	   ws2Idx  = (warpRow * strideWS2) 
		          + (WarpSize >> 1u)
		          + warpCol;
	}


	//------------------------------------
	// Zero out 'arrays'
	//------------------------------------

   uint * setPtr = NULL;

	//-
	// Zero out 'row starts' array
	//-

   //setPtr = (&s_rowStarts[0]);
   //SetArray_BlockSeq
   //   < 
   //      uint, BlockSize, nSafePassesCnts, 
   //      leftOverCnts, sizeCounts 
   //   >
   //   ( 
   //      setPtr, 0u
   //   );


   //-
	// Zero out 'Serial Scan' array
	//-

   setPtr = (&s_ss1[0]);
   SetArray_BlockSeq
      < 
         uint, BlockSize, nSafePassesSS1, 
         leftOverSS1, sizeSS1 
      >
      ( 
         setPtr, 0u
      );


   //-
	// Zero out 'Warp Scan' array
	//-

   setPtr = (&s_ws2[0]);
   SetArray_BlockSeq
      < 
         uint, BlockSize, nSafePassesWS2, 
         leftOverWS2, sizeWS2 
      >
      ( 
         setPtr, 0u
      );


   // Sync all threads in block
   if (WarpsPerBlock > 2u) { __syncthreads(); }


	//-------------------------------------------------
   // Phase 1:
	//   Serial Reduction of all rows of 'per row' counts
	//	  down to single set of 'total' counts
	//-------------------------------------------------

   {
      const uint * inPtr = &inRowCounts[threadIdx.x];

	   // Initialize 'Thread Sum' to identity value
	   tSum = 0;

	   // Loop over row counts
	   #pragma unroll
	   for (uint currPass = 0u; currPass < nRows; currPass++)
	   {		
		   // Grab count from global arrary
		   uint currCnt = inPtr[0];

		   // Accumulate 'per row' counts into a 'total' count
		   tSum = tSum + currCnt;

		   // Move to next set of 'row counts' to process
         inPtr += BlockSize;
	   }

	   // Store the 'total count's
	   outTotalCounts[threadIdx.x] = tSum;

	   // Also store 'total count's into 'Serial Scan' array
      s_ss1[storeIdx] = tSum;

      // Sync all threads in block
      if (WarpsPerBlock > 2u) { __syncthreads(); }
   }


	//--------------------------------------
   // Phase 2:
	//   convert 'total counts' into 'total starts'
   //   using prefix sum
   //--------------------------------------

   if (warpRow == 0)
   {
	   volatile uint * wsPtr = (uint *)&(s_ws2[0]);
   	
      uint * SS1_ptr = &s_ss1[ss1Idx];

		   // For higher performance, we use registers instead of shared memory
		   // Tradeoff - lots of register pressure (8 registers per thread)
      uint ss01, ss02, ss03, ss04;
      uint ss05, ss06, ss07, ss08;

      //-----
      // Serial Scan (on short sequence of 8 values)
      //-----

      // Grab short sequence of 8 values from ss1 array
      ss01 = SS1_ptr[0];
      ss02 = SS1_ptr[1];
      ss03 = SS1_ptr[2];
      ss04 = SS1_ptr[3];
      ss05 = SS1_ptr[4];
      ss06 = SS1_ptr[5];
      ss07 = SS1_ptr[6];
      ss08 = SS1_ptr[7];

      // Serial scan short sequence (in registers)
      //ss01 = <identity> + ss01;
      ss02 = ss01 + ss02;
      ss03 = ss02 + ss03;
      ss04 = ss03 + ss04;
      ss05 = ss04 + ss05;
      ss06 = ss05 + ss06;
      ss07 = ss06 + ss07;
      ss08 = ss07 + ss08;

      //-
      // Store final serial scan result into warp scan array
      //-

      uint wi = ws2Idx;
      tSum = ss08;
      wsPtr[wi] = tSum;

	   //-----
	   // Warp Scan (on 32 threads in parallel)
	   //-----

      wsPtr[wi] = tSum = wsPtr[wi -  1u] + tSum;
      wsPtr[wi] = tSum = wsPtr[wi -  2u] + tSum;
      wsPtr[wi] = tSum = wsPtr[wi -  4u] + tSum;
      wsPtr[wi] = tSum = wsPtr[wi -  8u] + tSum;
      wsPtr[wi] = tSum = wsPtr[wi - 16u] + tSum;


      //-----
      // Serial Update (on short sequence of 8 values)
      //-----

      //-
      // Grab update (prefix) value from Warp Array
      //-
         // Note:  Need to reach back 'one column' to get exclusive result
      uint prevWI = wi - 1u;
      tSum = wsPtr[prevWI];


      //-
      // Update each element short sequence with prefix (in registers)
      //-

      ss01 = tSum + ss01;
      ss02 = tSum + ss02;
      ss03 = tSum + ss03;
      ss04 = tSum + ss04;
      ss05 = tSum + ss05;
      ss06 = tSum + ss06;
      ss07 = tSum + ss07;
      ss08 = tSum + ss08;

      // Store 'prefix sum' results back in 'serial scan' array
      SS1_ptr[0] = ss01;
      SS1_ptr[1] = ss02;
      SS1_ptr[2] = ss03;
      SS1_ptr[3] = ss04;
      SS1_ptr[4] = ss05;
      SS1_ptr[5] = ss06;
      SS1_ptr[6] = ss07;
      SS1_ptr[7] = ss08;
   } // end warpRow == 0

   // Sync all threads in block
   if (WarpsPerBlock > 2u) { __syncthreads(); }


   //-----
   // Grab starting 'row start' (total sum) for this thread
   //    Note #1:  Need to 'reach back' one column for exclusive results
   //    Note #2:  This will result in an unavoidable '2-way' bank conflict
   //-----

   uint rowSum = s_ss1[prevIdx];

	// Store total starts (from previous column)
	outTotalStarts[threadIdx.x] = rowSum;

   // Sync all threads in block
   if (WarpsPerBlock > 2u) { __syncthreads(); }


	//-------------------------------------------------
   // Phase 3:
   //    Accumulate and write out 'per row' starts
	//-------------------------------------------------

   {
      const uint * inPtr  = &inRowCounts[threadIdx.x];
            uint * outPtr = &outRowStarts[threadIdx.x];

	   // Initialize 'Thread Sum' to identity value

	   // Loop over row counts
	   #pragma unroll
	   for (uint currPass = 0u; currPass < nRows; currPass++)
	   {		
		   // Read 'in' current count from global arrary
		   uint currCnt = inPtr[0];

         // Write 'out' current row sum to global array
         outPtr[0] = rowSum;

		   // Accumulate 'per row' count into running 'row sum' start
		   rowSum = rowSum + currCnt;

         //-
		   // Move to next row
         //-
         
         inPtr  += BlockSize;
         outPtr += BlockSize;
	   }
      // Sync all threads in block
      //if (WarpsPerBlock > 2u) { __syncthreads(); }
   }
}


////////////////////////////////////////////////////////////////////////////////
// Host interface to GPU TRISH histogram
////////////////////////////////////////////////////////////////////////////////


/*-----------------
  Local Defines
-----------------*/

// GTX 560M
//#define NUM_GPU_SMs (4u)

// TESLA 2050 (2070)
//#define NUM_GPU_SMs (14u)

// GTX 480
#define NUM_GPU_SMs (15u)

// GTX 580
//#define NUM_GPU_SMs (16u)


// Intermediate CUDA buffers
static uint * d_rowCounts = NULL;
static uint * d_rowStarts = NULL;
static uint * d_totalStarts = NULL;


//-----------------------------------------------
// Name:  initTrish256
// Desc:  Initialize intermediate GPU Buffers
//-----------------------------------------------

extern "C" 
void initTrish256(void)
{
	//------
	// Local Constants
	//------

	const uint nHistBins256  = 256u;
	const uint nGPU_SMs      = NUM_GPU_SMs;
	const uint nGPU_ConcurrentBlocks = 3u;
	const uint K1_GridSize   = nGPU_SMs * nGPU_ConcurrentBlocks;
	const uint K1_nRows      = K1_GridSize;
	const uint sizeRowCounts = K1_nRows * nHistBins256 * sizeof(uint);
	const uint sizeTotal     = nHistBins256 * sizeof(uint);

	// Create intermediate GPU buffers
    cutilSafeCall( hipMalloc( (void **)&d_rowCounts, sizeRowCounts ) );
    cutilSafeCall( hipMalloc( (void **)&d_rowStarts, sizeRowCounts ) );
    cutilSafeCall( hipMalloc( (void **)&d_totalStarts, sizeTotal ) );
}



//-----------------------------------------------
// Name:  closeTrish256
// Desc:  cleanup intermediate GPU buffers
//-----------------------------------------------

extern "C" 
void closeTrish256(void)
{
	// Destroy Intermediate GPU buffers
    cutilSafeCall( hipFree( d_totalStarts ) );
	cutilSafeCall( hipFree( d_rowStarts ) );
	cutilSafeCall( hipFree( d_rowCounts ) );
}



//---------------------------------------------------------
// Name:  histogramTrish256
// Desc:  CPU Wrapper function around GPU kernels 
//        for use in "histogram" demo  
//---------------------------------------------------------

extern "C" 
void histogramTrish256
(
	// Function Parameters
    uint *d_Histogram,	// OUT - Final 256-way histogram counts
    void *d_Data,		//  IN - input data to bin & count into histogram
    uint byteCount		//  In - length of input data array
)
{
	//-----
	// Local Constants=
	//-----


      // Note:  The best # of blocks for the TRISH algorithm appears to be
      //        The # of SM's on the card * the number of concurrent blocks.
      //        This is the mininum to effectively use all hardware resources effectively.
      // 
      // For Example:  On the following Fermi cards, the grid sizes for best performance would be ... 
      //  GTX 560M    = 12 =  4 * 3
      //  TELSA M2050 = 42 = 14 * 3
      //  GTX 480     = 45 = 15 * 3
      //  GTX 580     = 48 = 16 * 3

	const uint nGPU_SMs     = NUM_GPU_SMs;	// See #defines above
	const uint nGPU_ConcurrentBlocks = 3u;	// for Fermi architectures, we can achieve 3 concurrent blocks per SM (64 * 3 = 192 => 192/1536 => 12.5% occupancy 
	const uint logBankSize  = 5u;		//  5 = log<2>( Memory Banks )
	const uint logWarpSize  = 5u;       //  5 = log<2>( Threads per Warp )
	
	const uint K1_BlockSize = 64u;      // 64 = Threads per Block (Histogram Kernel)
	const uint K1_GridSize  = nGPU_SMs * nGPU_ConcurrentBlocks;	 // GridSize (Histogram Kernel)

	const uint K2_BlockSize = 256u;		// 256 = Threads per Block (RowSum Kernel)
	const uint K2_GridSize  = 1u;		//  1 = GridSize (RowSum Kernel)
	
	const uint K1_Length    = 31u;		//  31 = Work Per thread (loop unrolling)
	const uint in_start     = 0u;		//   0 = starting range
	const uint K1_nRows     = K1_GridSize;	//  ?? = Number of rows (blocks) that are cooperatively striding across input data set


	//-----
	// Get number of elements
	//-----

    assert( byteCount > 0u );
    assert( byteCount % sizeof(uint) == 0u );

	uint nElems = byteCount >> 2u;  // byteCount/4
	uint in_stop = nElems - 1u;	

	const uint * d_inVals = (const uint *)d_Data;


	/*--------------------------------------
	  Step 0. Create Intermediate buffers 
    --------------------------------------*/

	// Code moved to initTrish256() above


	/*------------------------------------------------------
	  Step 1. Bin & count elements into 'per row' 256-way histograms
	------------------------------------------------------*/

	H_K1_CountRows_256_TRISH
		< 
		  // Template Parameters
		  logBankSize,		// log<2>( Memory Banks ) 
		  logWarpSize,		// log<2>( Threads per Warp )
		  K1_BlockSize,		// Threads per Block
		  K1_GridSize,      // Blocks per Grid
		  K1_Length			// Work Per Thread (Loop unrolling)
		>
		<<< 
			// CUDA CTA Parameters
			K1_GridSize,	// Blocks per Grid 
			K1_BlockSize	// Threads per Block
		>>>
		(
			// Function parameters
			d_rowCounts,	// IN - 'per row' histograms
			d_inVals,		// IN - 'input' data to count & bin
			in_start,		// IN - input range [start, stop] 
			in_stop			//      ditto
		);
   // Check if kernel execution generated an error    
   cutilCheckMsg( "H_K1_CountRows_256() Kernel execution failed!" );


	/*------------------------------------------------------
	    Step 2. Sum 'per row' histograms into 'final' 256-bin histogram
	------------------------------------------------------*/

    H_K2_RowCounts_To_RowStarts_256
		< 
			// Template Parameters
			logBankSize,	// log<2>( Memory Banks ) 
			logWarpSize,	// log<2>( Warp Size )
			K2_BlockSize	// Threads per Block
		>
        <<< 
			// CUDA CTA Parameters
			K2_GridSize,	// Blocks per Grid 
			K2_BlockSize	// Threads per Block
		>>>	
        (
			// Function parameters
			d_Histogram,    // OUT - Histogram Counts
			d_totalStarts,  // OUT - Histogram Starts
			d_rowStarts,    // OUT - 'Per Row' Histogram Starts
			d_rowCounts,    // IN  - 'Per Row' Histogram Counts
			K1_nRows		// IN  - number of rows
        );
	// Check if kernel execution generated an error    
	cutilCheckMsg( "H_K2_RowCounts_To_RowStarts_256() Kernel execution failed!" );


	/*--------------------------------------
	  Step 3. Cleanup intermediate buffers
	--------------------------------------*/

	// Code moved to closeTrish256() above
}